#include "hip/hip_runtime.h"
#include <iostream>

#include "MapPoint.hpp"
#include "Polyhedron.cuh"
#include "Particle.hpp"
#include "fucking_shit.cuh"
#include "model_constants.hpp"
#include "random_generator.cuh"
namespace jc = jones_constants;
typedef long long ll;

#define force_one_threaded_kernel() if(threadIdx.x || threadIdx.y || threadIdx.z || \
        blockIdx.x || blockIdx.y || blockIdx.z) return


const ll cuda_block_size = 256;


__global__ void init_food(...)
{
    force_one_threaded_kernel();

    // <initialization here>
}

__global__ void init_polyhedron(Polyhedron *polyhedron, ...)
{
    force_one_threaded_kernel();

    /* WARNING!!! As you can see, we're creating a new `Polyhedron` object
     * and _copying_ it to `*polyhedron`, not assigning the pointer. This
     * is done in purpose, to make it possible to copy `*polyhedron` back to
     * host code.
     */
    *polyhedron = *(new Polyhedron(...));
}

__global__ void run_iteration(const Polyhedron *polyhedron, const ll *iteration_number)
{
    ll i = blockIdx.x * blockDim.x + threadIdx.x;

    if(jc::projectnutrients && *iteration_number >= jc::startprojecttime)
        // Projecting food:
        polyhedron->points[i].trail += polyhedron->points[i].food;

    // Diffuses trail in current point
    diffuse_trail(polyhedron, i);
    polyhedron->points[i].trail = polyhedron->points[i].temp_trail;

    if(polyhedron->points[i].contains_particle)
    {
        do_motor_behaviours(polyhedron->points, i);
        do_sensory_behaviours(polyhedron->points, i);

        if(jc::do_random_death_test && jc::death_random_probability > 0 &&
                *iteration_number > jc::startprojecttime)
            random_death_test(&polyhedron->points[i]);
        if(*iteration_number % jc::death_frequency_test == 0)
            death_test(&polyhedron->points[i]);
        if(*iteration_number % jc::division_frequency_test == 0)
            division_test(&polyhedron->points[i]);
    }
}

__host__ int main()
{
    // Initializing cuRAND:
    init_rand<<<1, 1>>>(time(nullptr));

    Polyhedron *polyhedron;
    hipMallocManaged((void **)&polyhedron, sizeof(Polyhedron));
    init_polyhedron<<<1, 1>>>(polyhedron);
    
    // <Precalculations (cos, sin, ...) here>
    init_food<<<1, 1>>>(...);
    
    ll *iteration_number;
    hipMallocManaged((void **)&iteration_number, sizeof(ll));

    const ll cuda_grid_size = (polyhedron->get_n_of_points() + cuda_block_size - 1) /
            cuda_block_size;
    for(*iteration_number = 0; ; ++*iteration_number)
    {
        run_iteration<<<cuda_grid_size, cuda_block_size>>>(polyhedron, iteration_number);
        // <redrawing here>
    }
}
