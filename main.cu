#include "hip/hip_runtime.h"
#include <iostream>

#include "MapPoint.hpp"
#include "SimulationMap.cuh"
#include "Particle.cuh"
#include "fucking_shit.cuh"
#include "model_constants.hpp"
#include "random_generator.cuh"

namespace jc = jones_constants;
typedef long long ll;

#define force_one_threaded_kernel() if(threadIdx.x || threadIdx.y || threadIdx.z || \
        blockIdx.x || blockIdx.y || blockIdx.z) return


const ll cuda_block_size = 256;


__global__ void init_food(...)
{
    force_one_threaded_kernel();

    // <initialization here>
}

__global__ void init_polyhedron(SimulationMap *polyhedron, ...)
{
    force_one_threaded_kernel();

    /* WARNING!!! As you can see, we're creating a new `SimulationMap` object
     * and _copying_ it to `*polyhedron`, not assigning the pointer. This
     * is done in purpose, to make it possible to copy `*polyhedron` back to
     * host code.
     */
    *polyhedron = *(new SimulationMap(...));
}

__global__ void run_iteration(const SimulationMap *polyhedron, ll *iteration_number)
{
    MapPoint *self = &polyhedron->points[blockIdx.x * blockDim.x + threadIdx.x];

    if(jc::projectnutrients && *iteration_number >= jc::startprojecttime)
        // Projecting food:
        self->trail += self->food;

    // Diffuses trail in current point
    diffuse_trail(self);

    if(self->contains_particle)
    {
        do_motor_behaviours(polyhedron, self);
        do_sensory_behaviours(polyhedron, self);

        if(jc::do_random_death_test && jc::death_random_probability > 0 &&
           *iteration_number > jc::startprojecttime)
            random_death_test(self);
        if(*iteration_number % jc::death_frequency_test == 0)
            death_test(self);
        if(*iteration_number % jc::division_frequency_test == 0)
            division_test(self);
    }

    ++*iteration_number;
}

__global__ void iteration_post_triggers(const SimulationMap *polyhedron)
{
    MapPoint *self = &polyhedron->points[blockIdx.x * blockDim.x + threadIdx.x];

    self->trail = self->temp_trail;
}

__host__ int main()
{
    // Initializing cuRAND:
    init_rand<<<1, 1>>>(time(nullptr));

    SimulationMap *polyhedron;
    hipMallocManaged((void **) &polyhedron, sizeof(SimulationMap));
    init_polyhedron<<<1, 1>>>(polyhedron);

    // <Precalculations (like cos, sin, ...) here>
    init_food<<<1, 1>>>(...);

    ll *iteration_number;
    hipMallocManaged((void **) &iteration_number, sizeof(ll));

    const ll cuda_grid_size = (polyhedron->get_n_of_points() + cuda_block_size - 1) /
                              cuda_block_size;
    for(*iteration_number = 0;; /* iteration_number is updated inside run_iteration,
                                  * because we're going to run this as a stream/graph later
                                  * and don't want cpu to do anything between runs within a group */)
    {
        run_iteration<<<cuda_grid_size, cuda_block_size>>>(polyhedron, iteration_number);
        iteration_post_triggers<<<cuda_grid_size, cuda_block_size>>>(polyhedron);
        // <redrawing here>
    }
}
