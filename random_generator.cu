#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"

#include "random_generator.h"

__device__ hiprandState_t state;

__global__ void init_rand(const unsigned long long seed)
{
    hiprand_init(seed, 0, 0, &state);
}

__device__ double rand01()
{
    return hiprand_uniform_double(&state);
}
