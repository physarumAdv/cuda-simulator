#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"

#include "random_generator.cuh"


__device__ hiprandState_t state;

__global__ void init_rand(unsigned long long seed)
{
    hiprand_init(seed, 0, 0, &state);
}

__device__ double rand01()
{
    return hiprand_uniform_double(&state);
}
