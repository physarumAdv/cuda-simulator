#include "hip/hip_runtime.h"
#include "MapNode.cuh"
#include "Particle.cuh"
#include "Polyhedron.cuh"


__device__ MapNode::MapNode(Polyhedron *const polyhedron, int polyhedron_face_id, SpacePoint coordinates) :
        polyhedron(polyhedron), trail(0), contains_food(false), coordinates(coordinates),
        polyhedron_face_id(polyhedron_face_id), left(nullptr), top(nullptr), right(nullptr), bottom(nullptr)
{}

__device__ MapNode::~MapNode()
{
    delete particle;
}


/**
 * Updates `MapNode`'s neighbor pointer with the given value
 *
 * If the given value is `nullptr` (neighbor is set already) or the `target`'s value is not `nullptr` (trying to set
 * "no neighbor"), nothing happens, otherwise the neighbor value is set
 *
 * @param target Pointer to the neighbor field
 * @param value Neighbor to be set
 *
 * @returns `true`, if the neighbor is updated, otherwise `false`
 */
__device__ inline bool set_neighbor(MapNode **target, MapNode *value)
{
    static_assert(sizeof(target) <= sizeof(unsigned long long *), "I think, I can't safely cast `MapNode **` to"
                                                                  "`unsigned long long *`");

    if(value == nullptr)
        return false;

    return nullptr == (MapNode *)atomicCAS((unsigned long long *)target, (unsigned long long)nullptr,
                                           (unsigned long long)value);
}


__device__ bool MapNode::set_left(MapNode *value)
{
    return set_neighbor(&left, value);
}

__device__ bool MapNode::set_top(MapNode *value)
{
    return set_neighbor(&top, value);
}

__device__ bool MapNode::set_right(MapNode *value)
{
    return set_neighbor(&right, value);
}

__device__ bool MapNode::set_bottom(MapNode *value)
{
    return set_neighbor(&bottom, value);
}


__device__ MapNode *MapNode::get_left() const
{
    return left;
}

__device__ MapNode *MapNode::get_top() const
{
    return top;
}

__device__ MapNode *MapNode::get_right() const
{
    return right;
}

__device__ MapNode *MapNode::get_bottom() const
{
    return bottom;
}


__device__ bool MapNode::contains_particle() const
{
    return particle != nullptr;
}

__device__ bool MapNode::attach_particle(Particle *p)
{
    static_assert(sizeof(&particle) <= sizeof(unsigned long long *), "I think, I can't safely cast `Particle **` to"
                                                                     "`unsigned long long *`");

    return nullptr == (Particle *)atomicCAS((unsigned long long *)&particle, (unsigned long long)nullptr,
                                            (unsigned long long)p);
}

__device__ Particle *MapNode::get_particle() const
{
    return particle;
}

__device__ void MapNode::detach_particle()
{
    particle = nullptr;
}

__device__ bool MapNode::detach_particle(Particle *p)
{
    static_assert(sizeof(&particle) <= sizeof(unsigned long long *), "I think, I can't safely cast `Particle **` to"
                                                                     "`unsigned long long *`");

    return p == (Particle *)atomicCAS((unsigned long long *)&particle, (unsigned long long)p,
                                      (unsigned long long)nullptr);
}
