#include "hip/hip_runtime.h"
#include <utility>

#include "MapNode.cuh"
#include "Particle.cuh"
#include "geometric/Polyhedron.cuh"


__host__ __device__ MapNode::MapNode(Polyhedron *polyhedron, int polyhedron_face_index, SpacePoint coordinates) :
        trail(0), temp_trail(0), left(nullptr), top(nullptr), right(nullptr), bottom(nullptr), polyhedron(polyhedron),
        polyhedron_face_index(polyhedron_face_index), coordinates(coordinates), contains_food(false), particle(nullptr)
{}

__host__ __device__ MapNode &MapNode::operator=(MapNode &&other) noexcept
{
    if(this != &other)
    {
        swap(polyhedron, other.polyhedron);
        swap(trail, other.trail);
        swap(temp_trail, other.temp_trail);
        swap(left, other.left);
        swap(top, other.top);
        swap(right, other.right);
        swap(bottom, other.bottom);
        swap(polyhedron_face_index, other.polyhedron_face_index);
        swap(coordinates, other.coordinates);
        swap(contains_food, other.contains_food);
        swap(particle, other.particle);
    }

    return *this;
}

__host__ __device__ MapNode::MapNode(MapNode &&other) noexcept
{
    particle = nullptr;

    *this = std::move(other);
}

__host__ __device__ MapNode::~MapNode()
{
    delete particle;
}


/**
 * Updates `MapNode`'s neighbor pointer with the given value
 *
 * If the given value is `nullptr` (neighbor is set already) or the `target`'s value is not `nullptr` (trying to set
 * "no neighbor"), nothing happens, otherwise the neighbor value is set
 *
 * @param target Pointer to the neighbor field
 * @param value Neighbor to be set
 *
 * @returns `true`, if the neighbor is updated, otherwise `false`
 *
 * @note This operation is thread-safe when compiled as CUDA code, thread-unsafe when compiled as C++
 */
__device__ inline bool set_mapnode_neighbor(MapNode **target, MapNode *value)
{
    static_assert(sizeof(target) <= sizeof(unsigned long long *), "I think, I can't safely cast `MapNode **` to "
                                                                  "`unsigned long long *`");

    if(value == nullptr)
        return false;

    return nullptr == (MapNode *)atomicCAS((unsigned long long *)target, (unsigned long long)nullptr,
                                           (unsigned long long)value);
}


__device__ bool MapNode::set_left(MapNode *value)
{
    return set_mapnode_neighbor(&left, value);
}

__device__ bool MapNode::set_top(MapNode *value)
{
    return set_mapnode_neighbor(&top, value);
}

__device__ bool MapNode::set_right(MapNode *value)
{
    return set_mapnode_neighbor(&right, value);
}

__device__ bool MapNode::set_bottom(MapNode *value)
{
    return set_mapnode_neighbor(&bottom, value);
}


__host__ __device__ MapNode *MapNode::get_left() const
{
    return left;
}

__host__ __device__ MapNode *MapNode::get_top() const
{
    return top;
}

__host__ __device__ MapNode *MapNode::get_right() const
{
    return right;
}

__host__ __device__ MapNode *MapNode::get_bottom() const
{
    return bottom;
}


__host__ __device__ SpacePoint MapNode::get_coordinates() const
{
    return coordinates;
}

__host__ __device__ Polyhedron *MapNode::get_polyhedron() const
{
    return polyhedron;
}

__host__ __device__ int MapNode::get_face_index() const
{
    return polyhedron_face_index;
}

__host__ __device__ Face *MapNode::get_face() const
{
    return get_polyhedron()->get_faces() + get_face_index();
}


__host__ __device__ bool MapNode::does_contain_food() const
{
    return contains_food;
}

__host__ __device__ bool MapNode::does_contain_particle() const
{
    return particle != nullptr;
}


[[nodiscard]] __device__ bool MapNode::attach_particle(Particle *p)
{
    static_assert(sizeof(&particle) <= sizeof(unsigned long long *), "I think, I can't safely cast `Particle **` to "
                                                                     "`unsigned long long *`");

    return nullptr == (Particle *)atomicCAS((unsigned long long *)&particle, (unsigned long long)nullptr,
                                            (unsigned long long)p);
}

__host__ __device__ Particle *MapNode::get_particle() const
{
    return particle;
}

__device__ void MapNode::detach_particle()
{
    particle = nullptr;
}

__device__ bool MapNode::detach_particle(Particle *p)
{
    static_assert(sizeof(&particle) <= sizeof(unsigned long long *), "I think, I can't safely cast `Particle **` to "
                                                                     "`unsigned long long *`");

    return p == (Particle *)atomicCAS((unsigned long long *)&particle, (unsigned long long)p,
                                      (unsigned long long)nullptr);
}


__host__ __device__ bool operator==(const MapNode &a, const MapNode &b)
{
    return a.coordinates == b.coordinates;
}


__host__ __device__ MapNode *find_nearest_mapnode_greedy(const SpacePoint &dest, MapNode *const start)
{
    MapNode *current = start;
    double current_dist = get_distance(dest, current->get_coordinates());
    while(true)
    {
        bool found_better = false;
        for(auto next : {current->get_left(), current->get_top(), current->get_right(), current->get_bottom()})
        {
            double next_dist = get_distance(dest, next->get_coordinates());
            if(next_dist < current_dist)
            {
                current = next;
                current_dist = next_dist;
                found_better = true;
                break;
            }
        }
        if(!found_better)
            break;
    }
    return current;
}

__host__ __device__ MapNode *find_nearest_mapnode(const Polyhedron *const polyhedron, const SpacePoint &dest,
                                                  MapNode *const start)
{
    Face *dest_face = polyhedron->find_face_by_point(dest);

    if(start != nullptr)
    {
        MapNode *ans = find_nearest_mapnode_greedy(dest, start);
        if(*ans->get_face() == *dest_face)
            return ans;
    }

    return find_nearest_mapnode_greedy(dest, dest_face->get_node());
}
