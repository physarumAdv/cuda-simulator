#include "hip/hip_runtime.h"
#include "Polyhedron.cuh"
#include "common.cuh"


__device__ Polyhedron::Polyhedron(Face *faces, int n_of_faces) :
        faces(malloc_and_copy(faces, n_of_faces)), n_of_faces(n_of_faces)
{

}

__device__ Polyhedron &Polyhedron::operator=(const Polyhedron &other)
{
    if(this != &other)
    {
        faces = malloc_and_copy(other.faces, other.n_of_faces);
        n_of_faces = other.n_of_faces;
    }
    return *this;
}

__device__ Polyhedron::Polyhedron(const Polyhedron &other)
{
    *this = other;
}

__device__ Polyhedron &Polyhedron::operator=(Polyhedron &&other) noexcept
{
    if(this != &other)
    {
        faces = nullptr;

        swap(faces, other.faces);
        swap(n_of_faces, other.n_of_faces);
    }

    return *this;
}

__device__ Polyhedron::Polyhedron(Polyhedron &&other) noexcept
{
    *this = std::move(other);
}

__device__ Polyhedron::~Polyhedron()
{
    free((void *)faces);
}


__device__ Face *Polyhedron::find_face_by_point(SpacePoint point) const
{
    for(int i = 0; i < n_of_faces; ++i)
    {
        Face *face = &faces[i];
        SpacePoint normal = (face->get_vertices()[1] - face->get_vertices()[0]) % (point - face->get_vertices()[0]);
        normal = normal / get_distance(normal, origin);
        if(normal * face->get_normal() >= 1 - eps)
            return face;
    }
    return &faces[0];
}

__device__ Face *Polyhedron::get_faces() const
{
    return faces;
}

__device__ int Polyhedron::get_n_of_faces() const
{
    return n_of_faces;
}


__device__ bool does_edge_belong_to_face(SpacePoint a, SpacePoint b, const Face *face)
{
    bool flag1 = false, flag2 = false;
    for(int i = 0; i < face->get_n_of_vertices(); ++i)
    {
        if(face->get_vertices()[i] == a)
            flag1 = true;
        if(face->get_vertices()[i] == b)
            flag2 = true;
    }
    return flag1 && flag2;
}

__device__ Face *find_face_next_to_edge(int vertex_id, Face *current_face, Polyhedron *polyhedron)
{
    for(int i = 0; i < polyhedron->get_n_of_faces(); ++i)
        if(polyhedron->get_faces()[i] != *current_face &&
           does_edge_belong_to_face(current_face->get_vertices()[vertex_id],
                                    current_face->get_vertices()[vertex_id + 1],
                                    &polyhedron->get_faces()[i]))
            return &polyhedron->get_faces()[i];
    return current_face;
}

__device__ SpacePoint find_intersection_with_edge(SpacePoint a, SpacePoint b, Face *current_face,
                                                  int *intersection_edge)
{
    for(int i = 0; i < current_face->get_n_of_vertices() - 1; ++i)
    {
        SpacePoint intersection = line_intersection(current_face->get_vertices()[i],
                                                    current_face->get_vertices()[i + 1], a, b);
        if(intersection != origin && is_in_segment(a, b, intersection) &&
           is_in_segment(current_face->get_vertices()[i], current_face->get_vertices()[i + 1], intersection) &&
           get_distance(intersection, a) > eps)
        {
            if(intersection_edge != nullptr)
            {
                *intersection_edge = i;
            }
            return intersection;
        }
    }
    return b;
}

__device__ SpacePoint get_projected_vector_end(SpacePoint a, SpacePoint b, Face *current_face, Polyhedron *polyhedron)
{
    int intersection_edge_vertex_id = 0;
    SpacePoint intersection = find_intersection_with_edge(a, b, current_face, &intersection_edge_vertex_id);

    SpacePoint normal_before = current_face->get_normal();
    SpacePoint normal_after = find_face_next_to_edge(intersection_edge_vertex_id, current_face,
                                                     polyhedron)->get_normal();
    SpacePoint moving_vector = (b - a) / get_distance(a, b);

    double phi_cos = normal_after * normal_before;
    double phi_sin = sin(acos(phi_cos));
    double alpha_cos = moving_vector * (normal_before % normal_after);

    SpacePoint faced_vector_direction = (normal_before + normal_after * phi_cos) * sin(acos(alpha_cos)) / phi_sin +
                                        (normal_before % normal_after) * alpha_cos / phi_sin;

    // If vector AB does not intersect any edge of face, `intersection` equals `b`,
    // so `faced_vector_direction` does not affect at all
    return intersection + faced_vector_direction * (get_distance(a, b) - get_distance(intersection, a));
}
