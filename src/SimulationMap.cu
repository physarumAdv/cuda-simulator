#include "hip/hip_runtime.h"
#include <cmath>

#include "SimulationMap.cuh"
#include "Polyhedron.cuh"
#include "Face.cuh"
#include "jones_constants.hpp"
#include "random_generator.cuh"


typedef bool(MapNode::*SetNodeMethod)(MapNode *);

typedef MapNode *(MapNode::*GetNodeMethod)() const;


namespace jc = jones_constants;


__device__ double const mapnode_dist = 2 * jc::speed;


__device__ SimulationMap::SimulationMap(Polyhedron *polyhedron) :
        polyhedron(polyhedron)
{
    bool create_new_nodes = true;  // New nodes are allowed to be created

    Face &start_face = polyhedron->faces[0];
    SpacePoint start_node_coordinates = (start_face.vertices[0] + start_face.vertices[1] +
                                         start_face.vertices[2]) / 3;
    nodes = (MapNode *)malloc(sizeof(MapNode));
    nodes[0] = MapNode(polyhedron, start_face.id, start_node_coordinates);
    n_of_nodes = 1;

    // Direction vector from first node to its top neighbor sets randomly
    SpacePoint direction_vector = relative_point_rotation(start_node_coordinates, start_face.vertices[0],
                                                          start_face.normal, M_PI * 2 * rand0to1());
    /*
     * Array of direction vectors from nodes with the same index
     * as in `SimulationMap::nodes` array to their top neighbors
     */
    auto *nodes_directions = (SpacePoint *)malloc(sizeof(SpacePoint));
    nodes_directions[0] = direction_vector * mapnode_dist / get_distance(direction_vector, origin);

    /*
     * Array of pointers to the `MapNode` member functions
     * Each of them returns the particular neighbor node
     * First array element corresponds to a top neighbor,
     * the following elements correspond to the following neighbors counterclockwise
     */
    GetNodeMethod get_node_neighbors[] = {
            &MapNode::get_top,
            &MapNode::get_left,
            &MapNode::get_bottom,
            &MapNode::get_right
    };

    /*
     * Array of pointers to the `MapNode` member functions
     * Each of them sets the link from current node to the particular neighbor node
     * First array element corresponds to a top neighbor,
     * the following elements correspond to the following neighbors counterclockwise
     */
    SetNodeMethod set_node_neighbors[] = {
            &MapNode::set_top,
            &MapNode::set_left,
            &MapNode::set_bottom,
            &MapNode::set_right
    };

    // Creating new nodes until it can be done, some nodes may have less neighbors than four
    for(int current_node_id = 0; current_node_id < n_of_nodes; ++current_node_id)
    {
        MapNode &current_node = nodes[current_node_id];
        double angle = 0;
        for(int i = 0; i < 4; ++i)
        {
            if((current_node.*get_node_neighbors[i])() == nullptr)
            {
                int neighbor_node_id = get_neighbor_node_id(current_node_id, &nodes_directions,
                                                            angle, create_new_nodes);
                if(neighbor_node_id != -1)
                {
                    (current_node.*set_node_neighbors[i])(&nodes[neighbor_node_id]);
                }
            }
            angle += M_PI_2;
        }

        if(current_node_id == n_of_nodes - 1)
        {
            /*
             * Starting from this moment: setting all pointers to neighbors that were not set earlier
             * The node that will be set as the neighbor is the closest node to hypothetical coordinates of neighbor
             */

            // Returns to the beginning of the `SimulationMap::nodes` array
            current_node_id = -1;

            // All nodes were created
            create_new_nodes = false;
        }
    }
}

__device__ SimulationMap::~SimulationMap()
{
    free(nodes);
}


__device__ SpacePoint SimulationMap::count_neighbor_node_coordinates(int current_node_id, SpacePoint top_direction,
                                                                     double angle, bool do_projection) const
{
    MapNode &current_node = nodes[current_node_id];
    int current_face_id = current_node.get_face_id();
    SpacePoint neighbor_coordinates = relative_point_rotation(current_node.get_coordinates(),
                                                              current_node.get_coordinates() + top_direction,
                                                              polyhedron->faces[current_face_id].normal, angle);
    if(do_projection)
    {
        neighbor_coordinates = get_projected_vector_end(current_node.get_coordinates(), neighbor_coordinates,
                                                        current_face_id, polyhedron);
    }
    return neighbor_coordinates;
}


__device__ int SimulationMap::find_index_of_nearest_node(SpacePoint dest) const
{
    int nearest_mapnode_id = 0;
    for(int neighbor = 0; neighbor < n_of_nodes; ++neighbor)
    {
        if(get_distance(nodes[neighbor].get_coordinates(), dest) <
           get_distance(nodes[nearest_mapnode_id].get_coordinates(), dest))
        {
            nearest_mapnode_id = neighbor;
        }
    }
    return nearest_mapnode_id;
}


__device__ void SimulationMap::set_direction_to_top_neighbor(int current_node_id, int neighbor_node_id,
                                                             SpacePoint **nodes_directions, double angle) const
{
    MapNode &neighbor_node = nodes[neighbor_node_id];
    MapNode &current_node = nodes[current_node_id];

    if(neighbor_node.get_face_id() == current_node.get_face_id())
    {
        (*nodes_directions)[neighbor_node_id] = (*nodes_directions)[current_node_id];
    }
    else
    {
        SpacePoint new_direction = neighbor_node.get_coordinates() -
                                   find_intersection_with_edge(current_node.get_coordinates(),
                                                               count_neighbor_node_coordinates(current_node_id,
                                                                                               (*nodes_directions)[current_node_id],
                                                                                               angle, false),
                                                               &polyhedron->faces[current_node.get_face_id()]);
        new_direction = relative_point_rotation(neighbor_node.get_coordinates(),
                                                neighbor_node.get_coordinates() + new_direction,
                                                polyhedron->faces[neighbor_node.get_face_id()].normal,
                                                -angle) -
                        neighbor_node.get_coordinates();
        (*nodes_directions)[neighbor_node_id] = new_direction * mapnode_dist / get_distance(new_direction, origin);
    }
}


__device__ int SimulationMap::get_neighbor_node_id(int current_node_id, SpacePoint **nodes_directions, double angle,
                                                   bool create_new_nodes)
{
    int current_face_id = nodes[current_node_id].get_face_id();

    // Hypothetical coordinates of neighbor node counted using direction to the top neighbor and `angle`
    SpacePoint neighbor_coordinates = count_neighbor_node_coordinates(current_node_id,
                                                                      (*nodes_directions)[current_node_id], angle,
                                                                      true);
    int next_face_id = polyhedron->find_face_id_by_point(neighbor_coordinates);
    int nearest_node_id = find_index_of_nearest_node(neighbor_coordinates);
    if(!create_new_nodes || (current_face_id == nodes[nearest_node_id].get_face_id() &&
                             get_distance(nodes[nearest_node_id].get_coordinates(), neighbor_coordinates) < eps))
    {
        // Neighbor node has already existed
        return nearest_node_id;
    }
    else if(current_face_id == next_face_id || polyhedron->faces[next_face_id].get_node() == nullptr)
    {
        // Neighbor node does not exist, but it can be created
        nodes = device_realloc(nodes, n_of_nodes, n_of_nodes + 1);
        *nodes_directions = device_realloc(*nodes_directions, n_of_nodes, n_of_nodes + 1);
        nodes[n_of_nodes] = MapNode(polyhedron, next_face_id, neighbor_coordinates);
        polyhedron->faces[next_face_id].set_node(&nodes[n_of_nodes], polyhedron);
        set_direction_to_top_neighbor(current_node_id, n_of_nodes, nodes_directions, angle);
        n_of_nodes++;
        return n_of_nodes - 1;
    }
    return -1;
}


__device__ int SimulationMap::get_n_of_nodes() const
{
    return this->n_of_nodes;
}

__global__ void get_n_of_nodes(const SimulationMap *const simulation_map, int *return_value)
{
    stop_all_threads_except_first;

    *return_value = simulation_map->get_n_of_nodes();
}

