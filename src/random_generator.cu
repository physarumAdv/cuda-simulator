#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "common.cuh"

#include "random_generator.cuh"


__device__ hiprandState_t state;

__global__ void init_rand(unsigned long long seed)
{
    STOP_ALL_THREADS_EXCEPT_FIRST

                hiprand_init(seed, 0, 0, &state);
}

__device__ double rand0to1()
{
    return hiprand_uniform_double(&state);
}
