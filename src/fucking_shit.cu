#include "hip/hip_runtime.h"
#ifdef COMPILE_FOR_CPU
#include <cmath>
#endif //COMPILE_FOR_CPU

#include <initializer_list>
#include <cstdio>

#include "fucking_shit.cuh"
#include "random_generator.cuh"
#include "jones_constants.hpp"
#include "Particle.cuh"
#include "jones_constants.hpp"

namespace jc = jones_constants;


[[nodiscard]] __device__ bool create_particle(MapNode *node)
{
    auto p = new Particle(node, rand0to1() * 2 * M_PI);

    if(node->attach_particle(p))
        return true;

    delete p;
    return false;
}

[[nodiscard]] __device__ bool delete_particle(MapNode *node)
{
    Particle *p = node->get_particle();

    if(!node->detach_particle(p))
        return false;

    delete p;
    return true;
}


__host__ __device__ void diffuse_trail(MapNode *node)
{
    auto left = node->get_left(), top = node->get_top(), right = node->get_right(), bottom = node->get_bottom();

    double sum = top->get_left()->trail + top->trail + top->get_right()->trail +
                 left->trail + node->trail + right->trail +
                 bottom->get_left()->trail + bottom->trail + bottom->get_right()->trail;

    node->temp_trail = (1 - jc::diffdamp) * (sum / 9.0);
}


__host__ __device__ int count_particles_in_node_window(MapNode *node, int window_size)
{
    for(int i = 0; i < window_size / 2; ++i)
        node = node->get_top()->get_left();

    MapNode *row = node;
    int ans = 0;
    for(int i = 0; i < window_size; ++i)
    {
        MapNode *cur = row;
        for(int j = 0; j < window_size; ++j)
        {
            if(cur->does_contain_particle())
                ++ans;
            cur = cur->get_right();
        }
        row = row->get_bottom();
    }

    return ans;
}


__device__ bool random_death_test(MapNode *node)
{
    if(rand0to1() < jc::random_death_probability)
    {
        if(!delete_particle(node))
        {
            // This is what called "undefined behaviour" in the docs :)
            printf("%s:%d - this line should never be reached", __FILE__, __LINE__);
            return false; // Particle was not removed
        }
        return true; // Particle was removed
    }
    return false; // Particle was not removed
}

__device__ bool death_test(MapNode *node)
{
    int particles_in_window = count_particles_in_node_window(node, jc::sw);
    if(jc::smin <= particles_in_window && particles_in_window <= jc::smax)
    {/* if in survival range, then stay alive */}
    else
    {
        if(!delete_particle(node))
        {
            // This is what called "undefined behaviour" in the docs :)
            printf("%s:%d - this line should never be reached", __FILE__, __LINE__);
            return false; // Particle was not removed
        }
        return true; // Particle was removed
    }
    return false; // Particle was not removed
}

__device__ void division_test(MapNode *node)
{
    int particle_window = count_particles_in_node_window(node, jc::gw);
    if(jc::gmin <= particle_window && particle_window <= jc::gmax)
    {
        if(rand0to1() <= jc::division_probability)
        {
            MapNode *row = node->get_top()->get_left();
            for(int i = 0; i < 3; ++i)
            {
                MapNode *cur = row;
                for(int j = 0; j < 3; ++j)
                {
                    if(create_particle(cur)) // If new particle was successfully created
                        return;
                    cur = cur->get_right();
                }
                row = row->get_bottom();
            }
        }
    }
}


__host__ __device__ MapNode *find_nearest_mapnode_greedy(const SpacePoint &dest, MapNode *const start)
{
    MapNode *current = start;
    double current_dist = get_distance(dest, current->get_coordinates());
    while(true)
    {
        bool found_better = false;
        for(auto next : {current->get_left(), current->get_top(), current->get_right(), current->get_bottom()})
        {
            double next_dist = get_distance(dest, next->get_coordinates());
            if(next_dist < current_dist)
            {
                current = next;
                current_dist = next_dist;
                found_better = true;
                break;
            }
        }
        if(!found_better)
            break;
    }
    return current;
}

__host__ __device__ MapNode *find_nearest_mapnode(const Polyhedron *const polyhedron, const SpacePoint &dest,
                                                  MapNode *const start)
{
    Face *dest_face = polyhedron->find_face_by_point(dest);

    if(start != nullptr)
    {
        MapNode *ans = find_nearest_mapnode_greedy(dest, start);
        if(*ans->get_face() == *dest_face)
            return ans;
    }

    return find_nearest_mapnode_greedy(dest, dest_face->get_node());
}

#ifdef COMPILE_FOR_CPU

base_atomic_type atomicCAS(base_atomic_type *address, const base_atomic_type compare, const base_atomic_type val)
{
    base_atomic_type ans = *address;

    if(*address == compare)
        *address = val;

    return ans;
}

base_atomic_type atomicAdd(base_atomic_type *address, const base_atomic_type value)
{
    base_atomic_type ans = *address;
    *address += value;
    return ans;
}

double atomicAdd(double *address, const double value)
{
    double ans = *address;
    *address += value;
    return ans;
}

#endif //COMPILE_FOR_CPU

// The following code is compied from https://stackoverflow.com/a/62094892/11248508 and modified (@kolayne can explain)
// `address` CANNOT be pointer to const, because we are trying to edit memory by it's source
__device__ bool atomicCAS(bool *const address, const bool compare, const bool val)
{
    static_assert(sizeof(base_atomic_type) > 1, "The local atomicCAS implementation won't work if `base_atomic_type"
                                                   "type size <= 1");
    static_assert((sizeof(base_atomic_type) - 1 & sizeof(base_atomic_type)) == 0,
            "The local atomicCAS implementation won't work if `base_atomic_type` type size is not a power of 2");


    auto address_num = (unsigned long long)address;
    unsigned pos = address_num & (sizeof(base_atomic_type) - 1);  // byte position within the `base_atomic_type`
  
    auto *address_of_extended = (base_atomic_type *)(address - pos);  // `base_atomic_type`-aligned address
    base_atomic_type old_extended = *address_of_extended, compare_extended, current_value_extended;

    bool current_value;

    do
    {
        current_value = (bool)(old_extended & ((0xFFU) << (8 * pos)));

        if(current_value != compare) // If we expected that bool to be different, then
            break; // stop trying to update it and just return it's current value

        compare_extended = old_extended;

        if(val)
            current_value_extended = old_extended | (1U << (8 * pos));
        else
            current_value_extended = old_extended & (~((0xFFU) << (8 * pos)));

        old_extended = atomicCAS(address_of_extended, compare_extended, current_value_extended);
    } while(compare_extended != old_extended);

    return current_value;
}

// I (Nikolay Nechaev, @kolayne) have no idea why the fuck the following only works with #else. If you're reading this
// and now why, PLEASE, contact me and tell me
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
// The following code is from https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions and
// slightly changed (@kolayne can explain)
__device__ double atomicAdd(double* address, double val)
{
    auto* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif // !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
